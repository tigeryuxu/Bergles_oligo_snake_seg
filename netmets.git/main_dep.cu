#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string>
#include <fstream>
#include <algorithm>

//OpenGL includes
#include <GL/glut.h>
#include <GL/freeglut.h>

//STIM includes
#include <stim/visualization/gl_network.h>
#include <stim/biomodels/network.h>
#include <stim/visualization/gl_aaboundingbox.h>
#include <stim/parser/arguments.h>
#include <stim/visualization/camera.h>

#ifdef __HIPCC__
//CUDA includes
#include <hip/hip_runtime.h>
#endif

//ANN includes
//#include <ANN/ANN.h>

//BOOST includes
#include <boost/tuple/tuple.hpp>

//visualization objects
stim::gl_aaboundingbox<float> bb;			//axis-aligned bounding box object
stim::camera cam;					//camera object

unsigned num_nets = 0;
stim::gl_network<float> GT;			//ground truth network
stim::gl_network<float> T;			//test network

//hard-coded parameters
float resample_rate = 0.5f;			//sample rate for the network (fraction of sigma used as the maximum sample rate)
float camera_factor = 1.2f;			//start point of the camera as a function of X and Y size
float orbit_factor = 0.01f;			//degrees per pixel used to orbit the camera

//mouse position tracking
int mouse_x;
int mouse_y;

//OpenGL objects
GLuint cmap_tex = 0;				//texture name for the color map

//sets an OpenGL viewport taking up the entire window
void glut_render_single_projection(){

	glMatrixMode(GL_PROJECTION);					//load the projection matrix for editing
	glLoadIdentity();								//start with the identity matrix
	int X = glutGet(GLUT_WINDOW_WIDTH);				//use the whole screen for rendering
	int Y = glutGet(GLUT_WINDOW_HEIGHT);
	glViewport(0, 0, X, Y);							//specify a viewport for the entire window
	float aspect = (float)X / (float)Y;				//calculate the aspect ratio
	gluPerspective(60, aspect, 0.1, 1000000);		//set up a perspective projection
}

//sets an OpenGL viewport taking up the left half of the window
void glut_render_left_projection(){

	glMatrixMode(GL_PROJECTION);					//load the projection matrix for editing
	glLoadIdentity();								//start with the identity matrix
	int X = glutGet(GLUT_WINDOW_WIDTH) / 2;			//only use half of the screen for the viewport
	int Y = glutGet(GLUT_WINDOW_HEIGHT);
	glViewport(0, 0, X, Y);							//specify the viewport on the left
	float aspect = (float)X / (float)Y;				//calculate the aspect ratio
	gluPerspective(60, aspect, 0.1, 1000000);		//set up a perspective projection
}

//sets an OpenGL viewport taking up the right half of the window
void glut_render_right_projection(){

	glMatrixMode(GL_PROJECTION);					//load the projection matrix for editing
	glLoadIdentity();								//start with the identity matrix
	int X = glutGet(GLUT_WINDOW_WIDTH) / 2;			//only use half of the screen for the viewport
	int Y = glutGet(GLUT_WINDOW_HEIGHT);
	glViewport(X, 0, X, Y);							//specify the viewport on the right
	float aspect = (float)X / (float)Y;				//calculate the aspect ratio
	gluPerspective(60, aspect, 0.1, 1000000);		//set up a perspective projection
}

void glut_render_modelview(){

	glMatrixMode(GL_MODELVIEW);						//load the modelview matrix for editing
	glLoadIdentity();								//start with the identity matrix
	stim::vec3<float> eye = cam.getPosition();		//get the camera position (eye point)
	stim::vec3<float> focus = cam.getLookAt();		//get the camera focal point
	stim::vec3<float> up = cam.getUp();				//get the camera "up" orientation

	gluLookAt(eye[0], eye[1], eye[2], focus[0], focus[1], focus[2], up[0], up[1], up[2]);	//set up the OpenGL camera
}

//draws the network(s)
void glut_render(void) {

	if(num_nets == 1){											//if a single network is loaded
		glut_render_single_projection();						//fill the entire viewport
		glut_render_modelview();								//set up the modelview matrix with camera details
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);		//clear the screen
		GT.glCenterline(GT.nmags() - 1);						//render the GT network (the only one loaded)
	}

	if(num_nets == 2){											//if two networks are loaded	

		glut_render_left_projection();							//set up a projection for the left half of the window
		glut_render_modelview();								//set up the modelview matrix using camera details
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);		//clear the screen

		glEnable(GL_TEXTURE_1D);										//enable texture mapping
		glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);		//texture map will be used as the network color
		glBindTexture(GL_TEXTURE_1D, cmap_tex);							//bind the Brewer texture map

		GT.glCenterline(GT.nmags() - 1);						//render the GT network

		glut_render_right_projection();							//set up a projection for the right half of the window
		glut_render_modelview();								//set up the modelview matrix using camera details
		T.glCenterline(T.nmags() - 1);							//render the T network

	}

	glutSwapBuffers();
}

// defines camera motion based on mouse dragging
void glut_motion(int x, int y){
	

	float theta = orbit_factor * (mouse_x - x);		//determine the number of degrees along the x-axis to rotate
	float phi = orbit_factor * (y - mouse_y);		//number of degrees along the y-axis to rotate

	cam.OrbitFocus(theta, phi);						//rotate the camera around the focal point

	mouse_x = x;									//update the mouse position
	mouse_y = y;
		
	glutPostRedisplay();							//re-draw the visualization
}

// sets the mouse position when clicked
void glut_mouse(int button, int state, int x, int y){
	mouse_x = x;
	mouse_y = y;
}

#define BREWER_CTRL_PTS 11							//number of control points in the Brewer map
void texture_initialize(){

	//define the colormap
	static float  brewer_map[BREWER_CTRL_PTS][3] = {			//generate a Brewer color map (blue to red)
		{0.192157f, 0.211765f, 0.584314f},
		{0.270588f, 0.458824f, 0.705882f},
		{0.454902f, 0.678431f, 0.819608f},
		{0.670588f, 0.85098f, 0.913725f},
		{0.878431f, 0.952941f, 0.972549f},
		{1.0f, 1.0f, 0.74902f},
		{0.996078f, 0.878431f, 0.564706f},
		{0.992157f, 0.682353f, 0.380392f},
		{0.956863f, 0.427451f, 0.262745f},
		{0.843137f, 0.188235f, 0.152941f},
		{0.647059f, 0.0f, 0.14902f}
	};

	glGenTextures(1, &cmap_tex);								//generate a texture map name
	glBindTexture(GL_TEXTURE_1D, cmap_tex);						//bind the texture map

	glTexParameteri(GL_TEXTURE_1D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);		//enable linear interpolation
	glTexParameteri(GL_TEXTURE_1D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_1D, GL_TEXTURE_WRAP_S, GL_CLAMP);			//clamp the values at the minimum and maximum
	glTexImage1D(GL_TEXTURE_1D, 0, 3, BREWER_CTRL_PTS, 0, GL_RGB, GL_FLOAT,	//upload the texture map to the GPU
					brewer_map);
}

//Initialize the OpenGL (GLUT) window, including starting resolution, callbacks, texture maps, and camera
void glut_initialize(){
	
	int myargc = 1;					//GLUT requires arguments, so create some bogus ones
	char* myargv[1];
	myargv [0]=strdup ("netmets");

	glutInit(&myargc, myargv);									//pass bogus arguments to glutInit()
	glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);	//generate a color buffer, depth buffer, and enable double buffering
	glutInitWindowPosition(100,100);							//set the initial window position
	glutInitWindowSize(320,320);								//set the initial window size
	glutCreateWindow("NetMets - STIM Lab, UH");					//set the dialog box title

	
	// register callback functions
	glutDisplayFunc(glut_render);			//function executed for rendering - renders networks
	glutMouseFunc(glut_mouse);				//executed on a mouse click - sets starting mouse positions for rotations
	glutMotionFunc(glut_motion);			//executed when the mouse is moved while a button is pressed

	texture_initialize();					//set up texture mapping (create texture maps, enable features)

	stim::vec3<float> c = bb.center();		//get the center of the network bounding box

	//place the camera along the z-axis at a distance determined by the network size along x and y
	cam.setPosition(c + stim::vec<float>(0, 0, camera_factor * std::max(bb.size()[0], bb.size()[1])));
	cam.LookAt(c[0], c[1], c[2]);						//look at the center of the network

	glClearColor(1, 1, 1, 1);
}

#ifdef __HIPCC__
void setdevice(int &device){
	int count;
	hipGetDeviceCount(&count);				// numbers of device that are available
	if(count < device + 1){
	std::cout<<"No such device available, please set another device"<<std::endl;
	exit(1);
	}
}
#else
void setdevice(int &device){
	device = -1;
}
#endif

//compare both networks and fill the networks with error information
void compare(float sigma, int device){

	GT = GT.compare(T, sigma, device);						//compare the ground truth to the test case - store errors in GT
    T = T.compare(GT, sigma, device);						//compare the test case to the ground truth - store errors in T

	//calculate the metrics
	float FPR = GT.average(0);						//calculate the metrics
	float FNR = T.average(0);
	
	std::cout << "FNR: " << FPR << std::endl;		//print false alarms and misses
	std::cout << "FPR: " << FNR << std::endl;
}

// writes features of the networks i.e average segment length, tortuosity, branching index, contraction, fractal dimension, number of end and branch points to a csv file
// Pranathi wrote this - saves network features to a CSV file
void features(std::string filename){
		double avgL_t, avgL_gt, avgT_t, avgT_gt, avgB_t, avgB_gt, avgC_t, avgC_gt, avgFD_t, avgFD_gt;
		unsigned int e_t, e_gt, b_gt, b_t;
		avgL_gt = GT.Lengths();
		avgT_gt = GT.Tortuosities();
		avgL_t = T.Lengths();
		avgT_t = T.Tortuosities();
		avgB_gt = GT.BranchingIndex();
		avgB_t = T.BranchingIndex();
		avgC_gt = GT.Contractions();
		avgFD_gt = GT.FractalDimensions();
		avgC_t = T.Contractions();
		avgFD_t = T.FractalDimensions();
		e_gt = GT.EndP();
		e_t = T.EndP();
		b_gt = GT.BranchP();
		b_t = T.BranchP();
		std::ofstream myfile;
		myfile.open (filename.c_str());
		myfile << "Length, Tortuosity, Contraction, Fractal Dimension, Branch Points, End points, Branching Index, \n";
		myfile << avgL_gt << "," << avgT_gt << "," << avgC_gt << "," << avgFD_gt << "," << b_gt << "," << e_gt << "," << avgB_gt <<std::endl;
		myfile << avgL_t << "," << avgT_t << "," << avgC_t << "," << avgFD_t << "," << b_t << "," << e_t << "," << avgB_t <<std::endl;
		myfile.close();
}

// Output an advertisement for the lab, authors, and usage information
void advertise(){
	std::cout<<std::endl<<std::endl;
	std::cout<<"========================================================================="<<std::endl;
	std::cout<<"Thank you for using the NetMets network comparison tool!"<<std::endl;
	std::cout<<"Scalable Tissue Imaging and Modeling (STIM) Lab, University of Houston"<<std::endl;
	std::cout<<"Developers: Pranathi Vemuri, David Mayerich"<<std::endl;
	std::cout<<"Source: https://git.stim.ee.uh.edu/segmentation/netmets"<<std::endl;
	std::cout<<"========================================================================="<<std::endl<<std::endl;

	std::cout<<"usage: netmets file1 file2 --sigma 3"<<std::endl;
	std::cout<<"            compare two files with a tolerance of 3 (units defined by the network)"<<std::endl<<std::endl;
	std::cout<<"       netmets file1 --gui"<<std::endl;
	std::cout<<"            load a file and display it using OpenGL"<<std::endl<<std::endl;
	std::cout<<"       netmets file1 file2 --device 0"<<std::endl;
	std::cout<<"            compare two files using device 0 (if there isn't a gpu, use cpu)"<<std::endl<<std::endl;
}

int main(int argc, char* argv[])
{
	stim::arglist args;						//create an instance of arglist

	//add arguments
	args.add("help", "prints this help");
	args.add("sigma", "force a sigma value to specify the tolerance of the network comparison", "3");
	args.add("gui", "display the network or network comparison using OpenGL");
	args.add("device", "choose specific device to run", "0");
	args.add("features", "save features to a CSV file, specify file name");

	args.parse(argc, argv);					//parse the user arguments

	if(args["help"].is_set() || args.nargs() == 0){			//test for help
		advertise();										//output the advertisement
		std::cout<<args.str();								//output arguments
		exit(1);											//exit
	}
	
	if(args.nargs() >= 1){					//if at least one network file is specified
		num_nets = 1;						//set the number of networks to one
		GT.load_obj(args.arg(0));			//load the specified file as the ground truth
		/*GT.to_txt("Graph.txt");*/
	}
	
	if(args.nargs() == 2){			//if two files are specified, they will be displayed in neighboring viewports and compared
		int device = args["device"].as_int();				//get the device value from the user
		num_nets = 2;										//set the number of networks to two
		float sigma = args["sigma"].as_float();				//get the sigma value from the user
		T.load_obj(args.arg(1));                           //load the second (test) network
		if(args["features"].is_set())						//if the user wants to save features
			features(args["features"].as_string());
		GT = GT.resample(resample_rate * sigma);			//resample both networks based on the sigma value
		T = T.resample(resample_rate * sigma);
		setdevice(device);
		compare(sigma, device);										//run the comparison algorithm
	}

	//if a GUI is requested, display the network using OpenGL
	if(args["gui"].is_set()){		
		bb = GT.boundingbox();					//generate a bounding volume		
		glut_initialize();						//create the GLUT window and set callback functions		
		glutMainLoop();							// enter GLUT event processing cycle
	}	
}
